#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "cudalab1.h"

int main(int argc, char **argv) 
{
	//CUDA lab 1 
	// Query the PC's NVIDIA GPU device and display some of it's characteristics.
	// Launch a simple kernel on the GPU.
	//**********************************************************************************

	
	printf("CUDA Lab 1\n\n");

	
	//Query the device properties. 
	//CUDA provides a hipDeviceProp_t structure which stores device properties.
	//We declare an in instance of the structure and pass it's address along with the GPU number (default 0) into the hipGetDeviceProperties function.
	//The function populates the structure properties with info.
	hipDeviceProp_t dprops;
	hipGetDeviceProperties(&dprops,0);
		
	
	//Output device name and number of MPs 
	printf("Found device %s, with %d multiprocessors.\n\n", 
		dprops.name, 
		dprops.multiProcessorCount );

	//There are many properties available to examine!
	// e.g.    	dprops.totalGlobalMem  ,  	dprops.maxThreadsPerBlock   etc etc

		
	//Define an array in the computers RAM
	int pcdata[1024];

	//Work out the number of elements in array and assign a number to each element
	//Useful C tip : Number of elements in array = total number of bytes in the array divided by number bytes per array element !
	int pcdataElements = sizeof(pcdata) / sizeof(pcdata[0]); 
	for(int i=0;i < pcdataElements; i++)
	{
		pcdata[i] = i;

		if(i<5)
			printf("%d ", pcdata[i]);
	}


	//Copy the array onto the GPU
	//First, define a pointer which will be the address of the memory alocated on the gpu
	int *gpudata;	

	//Allocate memory on the gpu using hipMalloc
	//hipMalloc expects a pointer and the number of bytes to allocate
	hipMalloc((void**)&gpudata, sizeof(pcdata));

	//Copy data onto the gpu at the appropriate address
	hipMemcpy(gpudata,pcdata,sizeof(pcdata),hipMemcpyHostToDevice);
	
	
	//Invoke the kernel.  It consists of a grid of 1 block which contains 1024 threads.
	SimpleKernel<<<1,1024>>>(gpudata);

	
	//Copy the results array back to the computers RAM
	hipMemcpy(pcdata,gpudata,sizeof(pcdata),hipMemcpyDeviceToHost);

	
	//Free up the device memory after we are finished with it.
	hipFree(gpudata);

	printf("\n");
	for(int i=0;i < 5; i++)
	{
		if(i<5)
			printf("%d ", pcdata[i]);
	}


}


//This simple kernel multiplies each element of the array by 10.
//In C we use pointers to pass array references into functions (see lecture 2).
__global__ void SimpleKernel(int *gpudata)
{
	//Notice how we don't now have a loop to process the array!

	//Get the unique thread id. threadIdx.x contains each thread's unique Id within the block.
	int tid = threadIdx.x;

	//Map the thread id to the array element for this thread and do our processing
	gpudata[tid] = gpudata[tid] * 10;
}