#include "hip/hip_runtime.h"
#include "vectoradd.h"
#include <time.h>


void main(int argc, char *argv[])
{
	//our program takes one input parameter.
	// vectoradd.exe 0 = do a cpu vector addition
	// vectoradd.exe 1 = do a gpu vector addition	
	// If you are debugging in visual studio, make sure you include your parameter - see project properties/debugging/CommandArguments.

	// C counts the program name itself as the first parameter so we need to check there are 2 parameters.
	if(argc != 2)
	{
		// No parameter supplied, end the program
		printf("\n\nUsage : 0 = cpu addition , 1 = gpu addition\n\n");
		return;
	}

	// Convert the parameter string to an integer using C atoi() function.
	int additionType = atoi(argv[1]);

	//make sure additionType has a default value of 0 or 1
	if(additionType != 0 && additionType != 1)
		additionType = 0;
		

	// clock_t is a C stopwatch structure that that helps us to time a process.
	clock_t begin,end;

	// Get the number of bytes required by each array = number of elements multiplied by the byte-size of each element
	unsigned int numbytes = VECTORSIZE * sizeof(int);
	
	// These arrays are far too big to declare in local stack memory, so we allocate them dynamically in the heap memory (i.e. DRAM).
	// Local arrays are usually stored in stack memory. Declaring array as "int a[VECTORSIZE]" would use up our stack memory and we would get a "stack overflow" error.
	// "malloc" allows us to allocate memory on the heap (i.e. dynamic RAM or DRAM). It returns a void pointer to the first byte which we cast to an "unsigned int" pointer.
	unsigned int *a = (unsigned int*)malloc(numbytes);	
	unsigned int *b = (unsigned int*)malloc(numbytes);			
	unsigned int *r = (unsigned int*)malloc(numbytes);	
	

	// Call a function to fill the vectors a and b with pseudo-random values
	fillvector(a,b);
		

	// Add vectors. 
	// We want to add vectors a and b and store the results of each element pair in vector r.
	if(additionType == 0)
	{
		//CPU addition
		printf("\nPerforming CPU vector addition\n");
		begin = clock();	// start timer
		addVectors(a,b,r);
		end = clock();		// stop timer
	}
	else
	{
		//GPU add vectors
		printf("\nPerforming GPU vector addition\n");

		// Allocate memory on the gpu for vectors a , b and r.
		unsigned int *ga, *gb, *gr;
		hipMalloc((void**)&ga, numbytes);
		hipMalloc((void**)&gb, numbytes);
		hipMalloc((void**)&gr, numbytes);

		// Copy vectors a and b over to the GPU
		hipMemcpy(ga, a, numbytes, hipMemcpyHostToDevice);
		hipMemcpy(gb, b, numbytes, hipMemcpyHostToDevice);

		begin = clock(); // Start the timer
		gpu_addVectors<<< 1024,1024  >>>(ga,gb,gr); // Call the kernel with a grid of 1048 blocks, each with 1024 threads
		end = clock(); // Stop the timer

		// Copy the results vector gr from the device back to the host
		hipMemcpy(r, gr, numbytes, hipMemcpyDeviceToHost);

		// We should free up the memory on the gpu when we are finished with it by using hipFree.
		hipFree(ga);
		hipFree(gb);
		hipFree(gr);

		//Rest the device to flush profile data (helps the Visual Profiler tool)
		hipDeviceReset();
	}
		
	printf("\nTime taken = %f\n",(double)(end - begin)/CLOCKS_PER_SEC); // output timer results


	// Output the results vector r
	printf("\n\nFinished!  Outputting some results...");
	for(int i=0;i<5;i++)
	{
		printf("\n%d \t %d,%d = %d",i,a[i],b[i],r[i]);
	}
	printf("\n...");
	for(int i=VECTORSIZE-5;i < VECTORSIZE;i++)
	{
		printf("\n%d \t %d,%d = %d",i,a[i],b[i],r[i]);
	}

	// We should always free up dynamically allocated memory when we are finished with it by using free()
	free(a);
	free(b);
	free(r);

}



// CPU add 2 vectors v1 and v2, place the results in "result" vector
void addVectors(unsigned int *v1, unsigned int *v2, unsigned int *result)
{
	for(unsigned int i=0;i < VECTORSIZE;i++)
	{
		result[i] = v1[i] + v2[i];
	}

}



// GPU kernel to add 2 vectors v1 and v2, and place the results in "result" vector
__global__ void gpu_addVectors(unsigned int *v1, unsigned int *v2, unsigned int *result)
{
	// Calculate the unique thread index
	unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

	
	// Ordinarily we could use the following line of code to do the work:  
	// result[tid] = v1[tid] + v2[tid];   // use the unique thread id to map this thread to a particular vector element to do its work

	// However if the number of threads we launch is less than the VECTORSIZE, only some of the vector elements will be processed!	
	// In our case we have only launched a grid of 1024 blocks of 1024 threads, so only approx 1m elements will be processed. 
	// nb - CUDA imposes a block size limit of 1024 threads , and a grid size of 65536 blocks.

	// This while loop helps us cope with vectors that are much bigger than 1m elements...
	// Each thread will perform additions on several unique elements 
	// One in the range 0 - 1m
	// The next in the range 1m - 2m
	// The next in the range 2m - 3m
	// ... and so on, until the VECTORSIZE is reached.
	while(tid < VECTORSIZE) { 

		result[tid] = v1[tid] + v2[tid]; 	// use the unique thread id to map this thread to a particular memory address to do its work

		tid += blockDim.x * gridDim.x;	// increase the thread id by the total number of threads (1024 x 1024).
	}


	// If comment out the while loop and just leave in the "result[tid] = v1[tid] + v2[tid];" , the output will show you that only
	// the first 1m elements of our vectors were processed. The later elements of the results array will show random (i.e. unitialized) values.
	// nb- Just because you allocate memory in C for arrays, it doesn't mean that it will initialize the elements to any particular value such as zero !

	// When you see unexpected large negative or exponential values in C variables, it usually means you are either reading unitialized values or your
	// are reading outsize of your array bounds!
}